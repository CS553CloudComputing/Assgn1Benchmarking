#include "hip/hip_runtime.h"
#include<stdio.h>

#define SIZE 10240
#define MEMSIZE 10000000
#define B 1024

struct arg
{
	long blockSize;
	int loopCnt;
	long itrn;
}arguments;

float MemoryBenchmarkWrite()
{
	char *a;
	char *d_a;

	hipEvent_t startT, stopT;
	float millisecs = 0;

	hipEventCreate(&startT);
	hipEventCreate(&stopT);

	a = (char *)malloc(MEMSIZE*sizeof(char));
	hipMalloc(&d_a, MEMSIZE*sizeof(char));

	hipEventRecord(startT);

	for (int i = 0; i < arguments.itrn; i++)
	{
		hipMemcpy(d_a + i * (arguments.blockSize), a + i * (arguments.blockSize), arguments.blockSize * sizeof(char), hipMemcpyHostToDevice);
	}

	hipEventRecord(stopT);

	hipEventSynchronize(stopT);

	hipEventElapsedTime(&millisecs, startT, stopT);

	printf("Time in milliseconds : %f \n", millisecs);

	return millisecs;

}

float MemoryBenchmarkRead()
{
	char *a;
	char *d_a;

	hipEvent_t startT, stopT;
	float millisecs = 0;

	hipEventCreate(&startT);
	hipEventCreate(&stopT);

	a = (char *)malloc(MEMSIZE*sizeof(char));
	hipMalloc(&d_a, MEMSIZE*sizeof(char));

	hipEventRecord(startT);

	for (int i = 0; i < arguments.itrn; i++)
	{
		hipMemcpy(a + i * (arguments.blockSize), d_a + i * (arguments.blockSize), arguments.blockSize * sizeof(char), hipMemcpyDeviceToHost);
	}

	hipEventRecord(stopT);

	hipEventSynchronize(stopT);

	hipEventElapsedTime(&millisecs, startT, stopT);

	printf("Time in milliseconds : %f \n", millisecs);

	return millisecs;

}

double ThroughputInGBps(float result, long blockSize, long itrn)
{
	return ((double)(blockSize*itrn) / (1000 * 1000* 1000)) / (((double)result / 1000));
}

void LatencyWrite()
{
	float result = MemoryBenchmarkWrite();
	printf("For Write :\nLatency : %.5g ms \t\t Throughput : %lf\tGBps\n", ((double)result / (double)arguments.itrn), ThroughputInGBps(result, arguments.blockSize, arguments.itrn));

}

void LatencyRead()
{
	float result = MemoryBenchmarkRead();
	printf("For Read :\nLatency : %.5g ms \t\t Throughput : %lf\tGBps\n", ((double)result / (double)arguments.itrn), ThroughputInGBps(result, arguments.blockSize, arguments.itrn));

}

__global__ void vectorADD(int *a, int *b, int *c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < SIZE)
	{
		a[i] = a[i] * 1 + 1 - 3 - 5 + 7 + 8 + 9;
		b[i] = b[i] * 1 + 1 - 3 - 5 + 7 + 8 + 9;
		c[i] = a[i] + b[i];
	}
}

__global__ void vectorFADD(float *a, float *b, float *c)
{
	int i = threadIdx.x; +blockIdx.x * blockDim.x;
	if (i < SIZE)
	{
		a[i] = a[i] + 1.23 + 2.03;
		b[i] = b[i] + 2.23 + 1.03;
		c[i] = a[i] + b[i];
		c[i] = c[i] * SIZE;
	}
}

void calcIOPs()
{
	int blockSize;
	int minGridSize;
	int threads;

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	unsigned long long iop, iops;

	hipEvent_t startT, stopT;
	float millisecs = 0;

	hipEventCreate(&startT);
	hipEventCreate(&stopT);


	a = (int *)malloc(SIZE * sizeof(int));
	b = (int *)malloc(SIZE * sizeof(int));
	c = (int *)malloc(SIZE * sizeof(int));

	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++)
	{
		a[i] = 2;
		b[i] = 3;
		c[i] = 1;
	}

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &threads, vectorADD, 0, SIZE);
	blockSize = (SIZE + threads - 1) / threads;

	hipEventRecord(startT);

	vectorADD << <blockSize, threads >> >(d_a, d_b, d_c);
	hipEventRecord(stopT);

	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stopT);

	hipEventElapsedTime(&millisecs, startT, stopT);

	printf("Time in milliseconds : %f \n", millisecs);
	iop = SIZE * 16 * 1000;
	iops = iop / millisecs;
	printf("IOPS : %lu\n", iops);
	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_c);
}

void calcFLOPs()
{
	int threads;
	int minGridSize;
	int blocksize;

	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	unsigned long long flop, flops;

	hipEvent_t startT, stopT;
	float millisecs = 0;

	hipEventCreate(&startT);
	hipEventCreate(&stopT);


	a = (float *)malloc(SIZE * sizeof(float));
	b = (float *)malloc(SIZE * sizeof(float));
	c = (float *)malloc(SIZE * sizeof(float));

	hipMalloc(&d_a, SIZE * sizeof(float));
	hipMalloc(&d_b, SIZE * sizeof(float));
	hipMalloc(&d_c, SIZE * sizeof(float));

	for (int i = 0; i < SIZE; i++)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(float), hipMemcpyHostToDevice);

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &threads, vectorFADD, 0, SIZE);
	blocksize = (SIZE + threads - 1) / threads;

	hipEventRecord(startT);

	vectorFADD << <blocksize, threads >> >(d_a, d_b, d_c);
	hipEventRecord(stopT);

	hipMemcpy(c, d_c, SIZE * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stopT);

	hipEventElapsedTime(&millisecs, startT, stopT);

	printf("Time in milliseconds : %f \n", millisecs);
	flop = SIZE * 6 * 1000;
	flops = flop / millisecs;
	printf("FLOPS : %lu\n", flops);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_c);

}

__global__ void vectorADDS(int *a, int *b, int *c)
{
	for (int i = 0; i < SIZE; i++)
	{
		a[i] = a[i] * 1 + 1 - 3 - 5 ;
		b[i] = b[i] * 1 + 1 - 3 - 5 ;
		c[i] = a[i] + b[i];
	}
}

__global__ void vectorFADDS(float *a, float *b, float *c)
{
	for (int i = 0; i < SIZE; i++)
	{
		//a[i] = a[i] + 1.23;
		//b[i] = b[i] + 2.23;
		c[i] = a[i] + b[i];
	}
}

void calcSTFLOPs()
{
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	unsigned long long flop, flops;

	hipEvent_t startT, stopT;
	float millisecs = 0;

	hipEventCreate(&startT);
	hipEventCreate(&stopT);


	a = (float *)malloc(SIZE * sizeof(float));
	b = (float *)malloc(SIZE * sizeof(float));
	c = (float *)malloc(SIZE * sizeof(float));

	hipMalloc(&d_a, SIZE * sizeof(float));
	hipMalloc(&d_b, SIZE * sizeof(float));
	hipMalloc(&d_c, SIZE * sizeof(float));

	for (int i = 0; i < SIZE; i++)
	{
		a[i] = 1.202;
		b[i] = 2.403;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(startT);

	vectorFADDS << <1,1 >> >(d_a, d_b, d_c);
	hipEventRecord(stopT);

	hipMemcpy(c, d_c, SIZE * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stopT);

	hipEventElapsedTime(&millisecs, startT, stopT);

	printf("Time in milliseconds : %f \n", millisecs);
	flop = SIZE * 1 *1000;
	flops = flop / millisecs;
	printf("FLOPS : %lu\n", flops);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_c);
}

void calcSTIOPs()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	unsigned long iop, iops;

	hipEvent_t startT, stopT;
	float millisecs = 0;

	hipEventCreate(&startT);
	hipEventCreate(&stopT);


	a = (int *)malloc(SIZE * sizeof(int));
	b = (int *)malloc(SIZE * sizeof(int));
	c = (int *)malloc(SIZE * sizeof(int));

	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++)
	{
		a[i] = 2;
		b[i] = 3;
		c[i] = 1;
	}

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(startT);

	vectorADD << <1, 1 >> >(d_a, d_b, d_c);
	hipEventRecord(stopT);

	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stopT);

	hipEventElapsedTime(&millisecs, startT, stopT);

	printf("Time in milliseconds : %f \n", millisecs);
	iop = SIZE * 9 *1000;
	//millisecs to secs
	iops = iop / millisecs;
	printf("IOPS : %lu\n", iops);
	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_c);
	hipFree(d_c);
}

int main(int argi, char *argc[])
{
	long blockSize = 0;
	char *operations;

	operations = argc[1];

	if (*operations == 's')
	{
		calcSTFLOPs();
		calcSTIOPs();
		
		return 0;
	}

	if (*operations == 'm')
	{
		calcIOPs();
		calcFLOPs();
		return 0;
	}

	blockSize = atol(argc[2]);

	arguments.blockSize = blockSize;
	long itrn = MEMSIZE/blockSize;
	arguments.itrn = itrn;

	if (*operations == 'w')
	{
		LatencyWrite();
		return 0;
	}
	if (*operations == 'r')
	{
		LatencyRead();
		return 0;
	}
	
	return 0;
}